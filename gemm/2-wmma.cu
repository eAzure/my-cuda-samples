/*
    using tensor core
    wmma api
*/
/*
    wmma api:
    template<typename Use, int M, int N, int K, typename T, typename Layout=void> class fragment;
        Use: matrix_a matrix_b accumulator
        M, N, K represent the size of the tile matrix_a's size: M * K
        T: half float int
        layout: row_major col_major
    void load_matrix_sync(fragment<...> &a, const T* mptr, unsigned ldm, layout_t layout);
        mptr: 256bit align point to mem
        ldm: stride the multiply of 16
        layout: accumulator need declare "mem_row_major"
    void store_matrix_sync(const T* mptr, fragment<...>&a, unsigned ldm, layout_t layout);
    void fill_fragment(fragment<...> &a, const T& value);
        value to all element in a
    void mam_sync(fragment<...> &d, fragment<...> &a, fragment<...> &b, fragment<...> &c, bool staf=false);
        d = a * b + c

*/
#include <hip/hip_runtime.h>
#include <mma.h>
#include <iostream>
#include <functional>

using namespace std;
using namespace nvcuda;

const int M = 1024;
const int N = 1024;
const int K = 1024;

#define WARP_SIZE 32

/* random init the data */
void random_init_data(half *data, size_t size) {
    for (size_t i=0;i<size;i++) {
        data[i] = 1.0 * rand()  / RAND_MAX;
    }
}

/* check data with cpu compute */
bool check_with_cpu(half *A, half *B, float *C,
                    int m, int n, int k) {
    for (int i=0;i<m;i++) {
        for (int j=0;j<n;j++) {
            float sum = 0.f;
            for (int h=0;h<k;h++) {
                sum += float(A[i*k+h]) * float(B[h*n+j]);
            }
            if (isnan(C[i * n + j])) {
                printf("C[%d][%d] is nan\n", i, j);
                return false;
            }
            // origin set 1e-5, but there may be some decline in accuracy.
            if (std::fabs(sum - C[i * n + j]) / std::fabs(sum) > 1e-4f) {
                printf("C[%d][%d] not match, %f vs %f\n", i, j, sum, C[i * n + j]);
                return false;
            }
        }
    }
    return true;
}

/* record the executing time and the throughput */
void record_time_throughput(const char *kernel_tag, const function<void()> &kernel,
                            int trial) {
    float sum_time = 0.f;
    for (int i=0;i<trial;i++) {
        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);

        hipEventRecord(start);
        kernel();
        hipEventRecord(end);
        hipEventSynchronize(end);

        hipError_t error = hipGetLastError();
        const char *error_info = hipGetErrorString(error);
        if (strlen(error_info) != 8) {
            printf("CUDA error: %s, happens in iter: %d of kernel: %s\n", error_info, i, kernel_tag);
        }

        float time_ms;
        hipEventElapsedTime(&time_ms, start, end);
        sum_time += time_ms;

        hipEventDestroy(start);
        hipEventDestroy(end);
    }
    float time = 1.0 * sum_time / trial;
    printf("Executing time: %f ms\n", time);
    /* compute the throughput */
    long workload = long(M) * N * K * 2;
    double gflops = (double(workload) / 1e9) / (double(time) / 1e3);
    printf("GFLOPS: %f\n", gflops);
}

/* wmma kernel */
/* naive implement */
template <int M_tile=16, int N_tile=16, int K_tile=16>
__global__ void naive_wmma_kernel(half *A, half *B, float *C, int M, int N, int K) {
    // warp number per dim
    int warp_number_dim_n = N / N_tile;
    int warp_number_dim_k = K / K_tile;
    // thread locate warp id
    int thread_in_warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    // transform thread_in_warp_id to 2-dim
    int thread_in_warp_m_id = thread_in_warp_id / warp_number_dim_n; // 将一维warp_id展为二维
    int thread_in_warp_n_id = thread_in_warp_id % warp_number_dim_n;

    wmma::fragment<wmma::matrix_a, M_tile, N_tile, K_tile, half, wmma::row_major> A_frag;
    wmma::fragment<wmma::matrix_b, M_tile, N_tile, K_tile, half, wmma::row_major> B_frag;
    wmma::fragment<wmma::accumulator, M_tile, N_tile, K_tile, float> C_frag;

    wmma::fill_fragment(C_frag, 0.0f);

    // locate the compute index in C
    float *C_store_index = C + thread_in_warp_m_id * M_tile *N + thread_in_warp_n_id * N_tile;

    // compute along k dim
    for (int kidx=0;kidx<warp_number_dim_k;kidx++) {
        half *A_load_index = A + thread_in_warp_m_id * M_tile * K + kidx * K_tile;
        half *B_load_index = B + kidx * K_tile * N + thread_in_warp_n_id * N_tile;

        wmma::load_matrix_sync(A_frag, A_load_index, K);
        wmma::load_matrix_sync(B_frag, B_load_index, N);

        wmma::mma_sync(C_frag, A_frag, B_frag, C_frag);
    }
    wmma::store_matrix_sync(C_store_index, C_frag, N, wmma::mem_row_major);
}

/* using shared memory */
template<int M_tile=16, int N_tile=16, int K_tile=16>
__global__ void shared_wmma_kernel(half *A, half *B, float *C, int M, int N, int K) {
    
}

int main() {
    /* host data */
    half *h_A, *h_B;
    float *h_C;
    hipHostMalloc(&h_A, M * K * sizeof(half), hipHostMallocDefault);
    hipHostMalloc(&h_B, N * K * sizeof(half), hipHostMallocDefault);
    hipHostMalloc(&h_C, M * N * sizeof(float), hipHostMallocDefault);
    random_init_data(h_A, M * K);
    random_init_data(h_B, N * K);

    /* device data */
    half *d_A, *d_B;
    float *d_C;
    hipMalloc(&d_A, M * K * sizeof(half));
    hipMalloc(&d_B, N * K * sizeof(half));
    hipMalloc(&d_C, M * N * sizeof(float));

    /* copy data from host to device */
    hipMemcpy(d_A, h_A, M * K * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * K * sizeof(half), hipMemcpyHostToDevice);

    /* call the kernel and record the time and the throughput */

    /* call the kernel */
    constexpr int M_tile = 16;
    constexpr int N_tile = 16;
    constexpr int K_tile = 16;
    int GRID_DIM, BLOCK_DIM;
    int number_of_warp = (M/M_tile) * (N/N_tile);
    int number_of_thread = number_of_warp * WARP_SIZE;
    int BLOCK_DIM_DEFAULT = 512;
    
    if(number_of_thread < BLOCK_DIM_DEFAULT){
        GRID_DIM = 1;
        BLOCK_DIM = number_of_thread;
    }else{
        GRID_DIM = number_of_thread % BLOCK_DIM_DEFAULT ? 
            number_of_thread / BLOCK_DIM_DEFAULT + 1 : number_of_thread / BLOCK_DIM_DEFAULT ;
        BLOCK_DIM = BLOCK_DIM_DEFAULT;
    }
    record_time_throughput("wmma_naive", [&](){
        naive_wmma_kernel<M_tile, N_tile, K_tile><<<GRID_DIM, BLOCK_DIM>>>(d_A, d_B, d_C, M, N, K);
    ;}, 100);

    /* copy result from device to host */
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    /* check the result data */
    bool result_check = check_with_cpu(h_A, h_B, h_C, M, N, K);
    printf("Check result: %s\n", result_check ? "OK" : "Failed");

    /* release the source */
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
